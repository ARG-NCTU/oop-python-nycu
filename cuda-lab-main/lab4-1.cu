
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;
using namespace chrono;

// Define execution parameters
const unsigned int N = 1 << 26;

const int threadsPerBlock = 256; // Number of threads per block
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; // Number of blocks per grid

__global__ void VectorAdd(const float *A, const float *B, float *C, int N) {
    // Calculate the index for the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {
        C[i] = A[i] + B[i]; // Perform vector addition
    }
}

int main() {
    size_t size = N * sizeof(float);
    cout << "Length of vector: " << N << endl;

    // Allocate memory for host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i / 1000.0f;
        h_B[i] = i * 2 / 1000.0f;
    }

    float *d_A, *d_B, *d_C;

    // Allocate memory on the device
    auto malloc_start = high_resolution_clock::now();
    // cudaMalloc((void **)&d_A, size);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    auto malloc_end = high_resolution_clock::now();

    // Copy data from host to device
    auto copy_htod_start = high_resolution_clock::now();
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    auto copy_htod_end = high_resolution_clock::now();

    // Launch the CUDA kernel for vector addition
    cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << endl;
    auto add_start = high_resolution_clock::now();
    VectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize(); // Wait for the kernel to complete
    auto add_end = high_resolution_clock::now();

    // Copy the result from device to host
    auto copy_dtoh_start = high_resolution_clock::now();
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    auto copy_dtoh_end = high_resolution_clock::now();

    // Print the elapsed time for various operations
    cout << "Time elapsed for memory allocation: " 
         << duration_cast<milliseconds>(malloc_end - malloc_start).count() << " ms" << endl;

    cout << "Time elapsed for data transfer from host to device: " 
         << duration_cast<milliseconds>(copy_htod_end - copy_htod_start).count() << " ms" << endl;

    cout << "Time elapsed for kernel execution: " 
         << duration_cast<milliseconds>(add_end - add_start).count() << " ms" << endl;

    cout << "Time elapsed for data transfer from device to host: " 
         << duration_cast<milliseconds>(copy_dtoh_end - copy_dtoh_start).count() << " ms" << endl;

    cout << "Total time elapsed: " 
         << duration_cast<milliseconds>(copy_dtoh_end - malloc_start).count() << " ms" << endl;

    // Verify the result
    for (int i = 0; i < N; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            cerr << "Error at index " << i << endl;
            exit(1);
        }
    }

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    cout << "Vector addition completed successfully!" << endl;
    return 0;
}
