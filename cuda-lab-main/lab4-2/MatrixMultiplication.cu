#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>

using namespace std;
using namespace chrono;

#define GetIndex(mtx, row, col) ((row) * (mtx).width + (col))
#define BLOCK_SIZE 16

typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Implement matrix multiplication on the CPU
bool MatMulCPU(const Matrix& A, const Matrix& B, Matrix& C) {
    if (A.width != B.height) {
        return false;
    }

    delete[] C.elements; // Free existing memory
    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height]();

    // Matrix multiplication
    for (int i = 0; i < A.height; ++i) {
        for (int j = 0; j < B.width; ++j) {
            float result = 0;
            for (int e = 0; e < A.width; e++) {
                result += A.elements[GetIndex(A, i, e)] * B.elements[GetIndex(B, e, j)];
            }
            C.elements[GetIndex(C, i, j)] = result;
        }
    }

    return true;
}

// Matrix multiplication kernel
_global_ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < C.height && col < C.width) {
        float result = 0;
        for (int e = 0; e < A.width; e++) {
            result += A.elements[GetIndex(A, row, e)] * B.elements[GetIndex(B, e, col)];
        }
        C.elements[GetIndex(C, row, col)] = result;
    }
}

// Matrix multiplication - Host code
bool MatMulGPU(const Matrix& A, const Matrix& B, Matrix& C) {
    if (A.width != B.height) {
        return false;
    }

    delete[] C.elements; // Free existing memory
    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height];

    Matrix d_A, d_B, d_C;
    d_A.width = A.width; d_A.height = A.height;
    d_B.width = B.width; d_B.height = B.height;
    d_C.width = C.width; d_C.height = C.height;

    size_t size_A = A.width * A.height * sizeof(float);
    size_t size_B = B.width * B.height * sizeof(float);
    size_t size_C = C.width * C.height * sizeof(float);

    hipMalloc(&d_A.elements, size_A);
    hipMemcpy(d_A.elements, A.elements, size_A, hipMemcpyHostToDevice);
    
    hipMalloc(&d_B.elements, size_B);
    hipMemcpy(d_B.elements, B.elements, size_B, hipMemcpyHostToDevice);
    
    hipMalloc(&d_C.elements, size_C);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                 (A.height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize(); // Wait for the kernel to finish

    hipMemcpy(C.elements, d_C.elements, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return true;
}

int main() {
    const int size = 1 << 11;

    const int N = size; 
    const int M = size;
    const int P = size;

    cout << "Matrix dimensions: " << N << " x " << M << " and " << M << " x " << P << endl;
    cout << "Allocating matrices..." << endl;

    Matrix A, B, result_CPU, result_GPU;
    A.width = N; A.height = M; A.elements = new float[N * M]();
    B.width = M; B.height = P; B.elements = new float[M * P]();

    for (int i = 0; i < N * M; ++i) {
        A.elements[i] = i % 100 / 1000.0f;
    }
    for (int i = 0; i < M * P; ++i) {
        B.elements[i] = i % 200 / 1000.0f;
    }

    cout << "Start matrix multiplication..." << endl;

    auto start_cpu = high_resolution_clock::now();
    MatMulCPU(A, B, result_CPU);
    auto end_cpu = high_resolution_clock::now();
    duration<float> duration_cpu = end_cpu - start_cpu;
    cout << "Matrix multiplication on CPU completed in " << duration_cpu.count() << " seconds" << endl;

    auto start_gpu = high_resolution_clock::now();
    MatMulGPU(A, B, result_GPU);
    auto end_gpu = high_resolution_clock::now();
    duration<float> duration_gpu = end_gpu - start_gpu;
    cout << "Matrix multiplication on GPU completed in " << duration_gpu.count() << " seconds" << endl;

    // Verify the result
    bool success = true;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
            if (abs(result_CPU.elements[GetIndex(result_CPU, i, j)] - result_GPU.elements[GetIndex(result_GPU, i, j)]) > 1e-5) {
                cout << "Results mismatch at index (" << i << ", " << j << "): " 
                     << result_CPU.elements[GetIndex(result_CPU, i, j)] << " != " << result_GPU.elements[GetIndex(result_GPU, i, j)] << endl;
                success = false;
            }
        }
    }

    if (success) {
        cout << "Matrix multiplication completed successfully!" << endl;
    } else {
        cout << "Matrix multiplication completed with errors!" << endl;
    }

    // Free allocated memory
    delete[] A.elements;
    delete[] B.elements;
    delete[] result_CPU.elements;
    delete[] result_GPU.elements;

    return 0;
}
